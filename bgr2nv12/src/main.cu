#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <iostream>
#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/imgproc.hpp>
#include <string>

__device__ float pixel(uint8_t* map, int x, int y, int width, int height,
                       int step, int offset) {
  return (float)map[y * (width * step) + x * step + offset];
}

__device__ float Rgb2Y(float r0, float g0, float b0) {
  return (0.257f * r0 + 0.504f * g0 + 0.098f * b0 + 16.0f);
}

__device__ float Rgb2U(float r0, float g0, float b0) {
  return (-0.148f * r0 - 0.291f * g0 + 0.439f * b0 + 128.0f);
}

__device__ float Rgb2V(float r0, float g0, float b0) {
  return (0.439f * r0 - 0.368f * g0 - 0.071f * b0 + 128.0f);
}

__global__ void BGR2NV12(uint8_t* bgr, uint8_t** nv12, int width, int height) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int x = tid % width;
  int y = tid / width;

  if (tid >= width * height) return;

  /*
   *  focus on left-top value
   *    O X O X O X
   *    X X X X X X
   *    O X O X O X
   *    X X X X X X
   *
   *  calc r00 g00 b00 -> y00, u00, v00
   *       r10 g10 b10 -> y10, u10, v10
   *       r01 g01 b01 -> y01, u01, v01
   *       r11 g11 b11 -> y11, u11, v11
   *
   *       --->  y = sum(Y) / 4
   *       --->  u = sum(U) / 4
   *       --->  v = sum(V) / 4
   *
   *  Standard NV12 |  Standard NV21
   *  YYYY          |  YYYY
   *  YYYY          |  YYYY
   *  UVUV          |  VUVU
   *
   */
  if (y % 2 == 0 && (x) % 2 == 0) {
    float r00 = pixel(bgr, x, y, width, height, 3, 2);
    float g00 = pixel(bgr, x, y, width, height, 3, 1);
    float b00 = pixel(bgr, x, y, width, height, 3, 0);

    float r01 = pixel(bgr, x + 1, y, width, height, 3, 2);
    float g01 = pixel(bgr, x + 1, y, width, height, 3, 1);
    float b01 = pixel(bgr, x + 1, y, width, height, 3, 0);

    float r11 = pixel(bgr, x + 1, y + 1, width, height, 3, 2);
    float g11 = pixel(bgr, x + 1, y + 1, width, height, 3, 1);
    float b11 = pixel(bgr, x + 1, y + 1, width, height, 3, 0);

    float r10 = pixel(bgr, x, y + 1, width, height, 3, 2);
    float g10 = pixel(bgr, x, y + 1, width, height, 3, 1);
    float b10 = pixel(bgr, x, y + 1, width, height, 3, 0);

    float y00 = Rgb2Y(r00, g00, b00);
    float y01 = Rgb2Y(r01, g01, b01);
    float y10 = Rgb2Y(r10, g10, b10);
    float y11 = Rgb2Y(r11, g11, b11);

    float u00 = Rgb2U(r00, g00, b00);
    float u01 = Rgb2U(r01, g01, b01);
    float u10 = Rgb2U(r10, g10, b10);
    float u11 = Rgb2U(r11, g11, b11);

    float v00 = Rgb2V(r00, g00, b00);
    float v01 = Rgb2V(r01, g01, b01);
    float v10 = Rgb2V(r10, g10, b10);
    float v11 = Rgb2V(r11, g11, b11);

    float u0 = (u00 + u01 + u10 + u11) * 0.25f;
    float v0 = (v00 + v01 + v10 + v11) * 0.25f;

    nv12[0][y * width + x] = (unsigned char)(y00 + 0.5f);
    nv12[0][y * width + x + 1] = (unsigned char)(y01 + 0.5f);
    nv12[0][(y + 1) * width + x] = (unsigned char)(y10 + 0.5f);
    nv12[0][(y + 1) * width + x + 1] = (unsigned char)(y11 + 0.5f);

    nv12[1][(y / 2) * width + x] = (unsigned char)(u0 + 0.5f);
    nv12[1][(y / 2) * width + x + 1] = (unsigned char)(v0 + 0.5f);
  }
}

void showNv12(uint8_t** nv12, int width, int height) {
  // Allocate memory on the CPU for the NV12 image
  int size = width * height * 3 / 2;
  uint8_t* nv12_cpu = new uint8_t[size];

  // Copy the NV12 image from the GPU to the CPU
  hipMemcpy(nv12_cpu, nv12[0], width * height, hipMemcpyDeviceToHost);
  hipMemcpy(nv12_cpu + width * height, nv12[1], height * width / 2,
             hipMemcpyDeviceToHost);

  cv::Mat BGR;
  cv::Mat NV12(height * 3 / 2, width, CV_8UC1, nv12_cpu);

  cv::namedWindow("NV12", cv::WINDOW_NORMAL);
  cv::imshow("NV12", NV12);
  cv::waitKey(0);

  cv::cvtColor(NV12, BGR, cv::COLOR_YUV2BGR_NV12);
  cv::namedWindow("BGR", cv::WINDOW_NORMAL);
  cv::imshow("BGR", BGR);
  cv::waitKey(0);
}

int main(int argc, char* args[]) {
  if (argc < 2) {
    std::cout << "parameter error ,run the program again" << std::endl;
    return 0;
  }

  std::string input(args[1]);

  cv::Mat ORG = cv::imread(input);

  if (ORG.empty()) {
    std::cout << "Can't read image!" << std::endl;
    return -1;
  }

  cv::Size imageSize = ORG.size();
  int width = imageSize.width;
  int height = imageSize.height;
  int channel = ORG.channels();

  uint8_t* src;
  uint8_t** nv12;
  hipMallocManaged(&nv12, sizeof(uint8_t*) * 2);
  hipMallocManaged(&src, width * height * 3 * sizeof(uint8_t));
  hipMallocManaged(&nv12[0], width * height * sizeof(uint8_t));
  hipMallocManaged(&nv12[1], (height / 2) * width * sizeof(uint8_t));

  hipMemcpy(src, ORG.data, width * height * 3 * sizeof(uint8_t),
             hipMemcpyHostToDevice);

  int TPB = 128;
  int blocksize = ((width * height) + TPB - 1) / TPB;
  BGR2NV12<<<blocksize, TPB>>>(src, nv12, width, height);
  hipDeviceSynchronize();

  cv::namedWindow("ORG", cv::WINDOW_NORMAL);
  cv::imshow("ORG", ORG);
  cv::waitKey(0);

  showNv12(nv12, width, height);

  cv::destroyAllWindows();

  return 1;
}
